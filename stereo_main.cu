#include <stdio.h>
#include "libpng_wrapper.h"
#include "stereo_cpu.h"
#include "stereo_cuda.h"
#include "stereo_cuda_shared.h"


void testMiddleBuryCPU() {
    char s1[] = "/playpen2/jisan/workspace/Datasets/Middlebury/Art/view1.png";
	Image img1;
	readPNGFile(img1, s1);
	char s2[] = "/playpen2/jisan/workspace/Datasets/Middlebury/Art/view5.png";
	Image img2;
	readPNGFile(img2, s2);
	img1.displayStats();
	img2.displayStats();

	char s3[] = "/playpen2/jisan/workspace/Datasets/Middlebury/Art/out_cpu.png";
	BinaryStereoCPU binStereo(&img1, &img2, 7);
	DoubleImage res;
	binStereo.computeStereo(res);
    writePNGFile(res, s3);

	printf("Done\n");
}


unsigned char* get_flattened_color_array(int height, int width) {
    unsigned char* arr;
    int n = height * width * 3;
    hipMallocManaged(&arr, n * sizeof(unsigned char));
    return arr;
}


unsigned char* get_flattened_color_array(Image &img) {
    int n = img.height * img.width * 3;
    unsigned char* arrCPU = new unsigned char[n];
    unsigned char* arrGPU;
    hipMalloc(&arrGPU, sizeof(unsigned char) * n);
    int k = 0;
    for (int i = 0; i < img.height; i++) {
        for (int j = 0; j < img.width; j++) {
            for (int channel = 0; channel < 3; channel++)
                arrCPU[k++] = img.img[i][j][channel];
        }
    }
    hipMemcpy(arrGPU, arrCPU, sizeof(unsigned char) * n, hipMemcpyHostToDevice);
    delete[] arrCPU;
    return arrGPU;
}


void stereoGPU(char* img1Path, char* img2Path, char* resultPath) {
    Image img1, img2;
    readPNGFile(img1, img1Path);
    readPNGFile(img2, img2Path);
    int height = img1.height;
    int width = img1.width;
    unsigned char* colors1 = get_flattened_color_array(img1);
	unsigned char* colors2 = get_flattened_color_array(img2);
//	float* res = computeDisparityMap(colors1, colors2, height, width, 7);
	float* res = computeDisparityMapShared(colors1, colors2, height, width);
    DoubleImage resImg;
	resImg.init(height, width);
	int k = 0;
	for (int i = 0; i < height; i++) {
	    for (int j = 0; j < width; j++, k++)
	        resImg.img[i][j] = res[k];
	}
	hipFree(colors1);
    hipFree(colors2);
    delete[] res;
    writePNGFile(resImg, resultPath);
}



void testMiddleBuryGPU() {
    char s1[] = "/playpen2/jisan/workspace/Datasets/Middlebury/Art/view1.png";
	char s2[] = "/playpen2/jisan/workspace/Datasets/Middlebury/Art/view5.png";
	char s3[] = "/playpen2/jisan/workspace/Datasets/Middlebury/Art/out_gpu.png";
	stereoGPU(s1, s2, s3);
}


int main() {
//    testMiddleBuryCPU();
    testMiddleBuryGPU();
	return 0;
}
